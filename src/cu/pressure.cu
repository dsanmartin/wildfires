#include "hip/hip_runtime.h"
/**
 * @file pressure.c
 * @author Daniel San Martin (dsanmartinreyes@gmail.com)
 * @brief Functions for solving the Poisson equation.
 * @version 0.1
 * @date 2024-07-21
 * 
 * @copyright Copyright (c) 2024
 * 
 */

#include "../../include/cu/pressure.cuh"

__global__
void thomas_algorithm(double *a, double *b, double *c, hipfftDoubleComplex *d, hipfftDoubleComplex *x, hipfftDoubleComplex *l, hipfftDoubleComplex *u, hipfftDoubleComplex *y, Parameters parameters) {
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    int size = (Nx - 1) * (Ny - 1);
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    for (int ij = idx; ij < size; ij += stride) {
        int i = ij / (Ny - 1);
        int j = ij % (Ny - 1);
        u[IDX(i, j, 0, Nx - 1, Ny - 1, Nz - 1)] = make_hipDoubleComplex(b[IDX(i, j, 0, Nx - 1, Ny - 1, Nz - 1)], 0.0);
        for (int k = 1; k < Nz - 1; k++) {
            l[IDX(i, j, k - 1, Nx - 1, Ny - 1, Nz - 2)] = hipCdiv(make_hipDoubleComplex(a[IDX(i, j, k - 1, Nx - 1, Ny - 1, Nz - 2)], 0.0), u[IDX(i, j, k - 1, Nx - 1, Ny - 1, Nz - 1)]); 
            u[IDX(i, j, k, Nx - 1, Ny - 1, Nz - 1)] = hipCsub(make_hipDoubleComplex(b[IDX(i, j, k, Nx - 1, Ny - 1, Nz - 1)], 0.0), hipCmul(l[IDX(i, j, k - 1, Nx - 1, Ny - 1, Nz - 2)], make_hipDoubleComplex(c[IDX(i, j, k - 1, Nx - 1, Ny - 1, Nz - 2)], 0.0)));
        }
        y[IDX(i, j, 0, Nx - 1, Ny - 1, Nz - 1)] = d[IDX(i, j, 0, Nx - 1, Ny - 1, Nz - 1)];
        for (int k = 1; k < Nz - 1; k++) {
            y[IDX(i, j, k, Nx - 1, Ny - 1, Nz - 1)] = hipCsub(d[IDX(i, j, k, Nx - 1, Ny - 1, Nz - 1)], hipCmul(l[IDX(i, j, k - 1, Nx - 1, Ny - 1, Nz - 2)], y[IDX(i, j, k - 1, Nx - 1, Ny - 1, Nz - 1)]));
        }
        x[FFTWIDX(i, j, Nz - 2, Nx - 1, Ny - 1, Nz - 1)] = hipCdiv(y[IDX(i, j, Nz - 2, Nx - 1, Ny - 1, Nz - 1)], u[IDX(i, j, Nz - 2, Nx - 1, Ny - 1, Nz - 1)]);
        for (int k = Nz - 3; k >= 0; k--) {
            x[FFTWIDX(i, j, k, Nx - 1, Ny - 1, Nz - 1)] = hipCdiv(hipCsub(y[IDX(i, j, k, Nx - 1, Ny - 1, Nz - 1)], hipCmul(make_hipDoubleComplex(c[IDX(i, j, k, Nx - 1, Ny - 1, Nz - 2)], 0.0), x[FFTWIDX(i, j, k + 1, Nx - 1, Ny - 1, Nz - 1)])), u[IDX(i, j, k, Nx - 1, Ny - 1, Nz - 1)]);
        }
    }
}

__global__
void gammas_and_coefficients(double *kx, double *ky, double *gamma, double *a, double *b, double *c, Parameters parameters) {
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    double dz = parameters.dz;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    int size = (Nx - 1) * (Ny - 1) * (Nz - 2);
    for (int rsk = idx; rsk < size; rsk += stride) {
        int r = rsk / ((Ny - 1) * (Nz - 2));
        int s = (rsk % ((Ny - 1) * (Nz - 2))) / (Nz - 2);
        int k = rsk % (Nz - 2);
        if (k == 0) { // Use k = 0 to fill gamma matrix and first coefficients of a, b and c
            // gamma[FFTWIDX(r, s, k, Nx - 1, Ny - 1, 0)] = -2 - kx[r] * kx[r] - ky[s] * ky[s];
            gamma[IDX(r, s, 0, Nx - 1, Ny - 1, 1)] = -2 - kx[r] * kx[r] - ky[s] * ky[s];
            a[IDX(r, s, k, Nx - 1, Ny - 1, Nz - 2)] = 1.0 / (dz * dz), 0.0; 
            b[IDX(r, s, k, Nx - 1, Ny - 1, Nz - 1)] = -1.0 / dz, 0.0;
            // c[IDX(r, s, k, Nx - 1, Ny - 1, Nz - 2)] = (2.0 + 0.5 * gamma[FFTWIDX(r, s, k, Nx - 1, Ny - 1, 0)]) / dz;
            c[IDX(r, s, k, Nx - 1, Ny - 1, Nz - 2)] = (2.0 + 0.5 * gamma[IDX(r, s, 0, Nx - 1, Ny - 1, 1)]) / dz;
        } else { // The rest of the coefficients a and c
            a[IDX(r, s, k, Nx - 1, Ny - 1, Nz - 2)] = 1.0 / (dz * dz); 
            c[IDX(r, s, k, Nx - 1, Ny - 1, Nz - 2)] = 1.0 / (dz * dz);
        }
    }
}

__global__
void compute_f(double *U, hipfftDoubleComplex *f_in, hipfftDoubleComplex *p_top_in, double *p, Parameters parameters) {
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    int u_index = parameters.field_indexes.u;
    int v_index = parameters.field_indexes.v;
    int w_index = parameters.field_indexes.w;
    int im1, ip1, jm1, jp1;
    double dx = parameters.dx;
    double dy = parameters.dy;
    double dz = parameters.dz;
    double dt = parameters.dt;
    double rho_inf = parameters.rho_inf;
    double u_ijk, u_ip1jk, u_im1jk, u_iphjk, u_imhjk;
    double v_ijk, v_ijp1k, v_ijm1k, v_ijphk, v_ijmhk;
    double w_ijk, w_ijkp1, w_ijkm1, w_ijkp2, w_ijkm2, w_ijkph, w_ijkmh;
    double ux, vy, wz, f;
    // Loop over nodes to compute f
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    int size = Nx * Ny * Nz;
    for (int ijk = idx; ijk < size; ijk += stride) {
        int i = ijk / (Ny * Nz);
        int j = (ijk % (Ny * Nz)) / Nz;
        int k = ijk % Nz;               
        // Indexes for periodic boundary conditions
        im1 = (i - 1 + Nx - 1) % (Nx - 1);
        jm1 = (j - 1 + Ny - 1) % (Ny - 1);
        ip1 = (i + 1) % (Nx - 1);
        jp1 = (j + 1) % (Ny - 1);                
        // Local nodes
        u_ijk = U[u_index + IDX(i, j, k, Nx, Ny, Nz)];
        v_ijk = U[v_index + IDX(i, j, k, Nx, Ny, Nz)];
        w_ijk = U[w_index + IDX(i, j, k, Nx, Ny, Nz)];
        // Periodic boundary conditions on xy
        u_im1jk = U[u_index + IDX(im1, j, k, Nx, Ny, Nz)];
        u_ip1jk = U[u_index + IDX(ip1, j, k, Nx, Ny, Nz)];
        v_ijm1k = U[v_index + IDX(i, jm1, k, Nx, Ny, Nz)];
        v_ijp1k = U[v_index + IDX(i, jp1, k, Nx, Ny, Nz)];
        // dw/dz 
        if (k == 0) { // Bottom boundary                    
            w_ijkp1 = U[w_index + IDX(i, j, k + 1, Nx, Ny, Nz)];
            w_ijkp2 = U[w_index + IDX(i, j, k + 2, Nx, Ny, Nz)];
            wz = (-3 * w_ijk + 4 * w_ijkp1 - w_ijkp2) / (2 * dz); // dw/dz at z = z_min
        } else if (k == Nz - 1) { // Top boundary
            w_ijkm1 = U[w_index + IDX(i, j, k - 1, Nx, Ny, Nz)];
            w_ijkm2 = U[w_index + IDX(i, j, k - 2, Nx, Ny, Nz)];
            wz = (3 * w_ijk - 4 * w_ijkm1 + w_ijkm2) / (2 * dz); // dw/dz at z = z_max
        } else { // Interior
            w_ijkp1 = U[w_index + IDX(i, j, k + 1, Nx, Ny, Nz)];
            w_ijkm1 = U[w_index + IDX(i, j, k - 1, Nx, Ny, Nz)];
            w_ijkph = 0.5 * (w_ijk + w_ijkp1);
            w_ijkmh = 0.5 * (w_ijk + w_ijkm1);
            wz = (w_ijkph - w_ijkmh) / dz; // dw/dz at z = z_k
        }
        // Half derivatives
        u_iphjk = 0.5 * (u_ip1jk + u_ijk);
        u_imhjk = 0.5 * (u_ijk + u_im1jk);
        v_ijphk = 0.5 * (v_ijp1k + v_ijk);
        v_ijmhk = 0.5 * (v_ijk + v_ijm1k);
        ux = (u_iphjk - u_imhjk) / dx; // du/dx
        vy = (v_ijphk - v_ijmhk) / dy; // dv/dy
        if (i < Nx - 1 && j < Ny - 1 && k < Nz - 1) {
            // Compute rho / dt * div(U) and store it for many DFT (contiguous z slices)
            f = rho_inf * (ux + vy + wz) / dt;
            f_in[FFTWIDX(i, j, k, Nx - 1, Ny - 1, Nz - 1)] = make_hipDoubleComplex(f, 0.0);
        }
        // Fill p_top
        if (k == Nz - 1 && j < Ny - 1 && i < Nx - 1) {
            p_top_in[FFTWIDX(i, j, 0, Nx - 1, Ny - 1, Nz - 1)] = make_hipDoubleComplex(p[IDX(i, j, k, Nx, Ny, Nz)], 0.0);
        }
    }
}

__global__
void compute_f_density(double *U, hipfftDoubleComplex *f_in, hipfftDoubleComplex *p_top_in, double *p, Parameters parameters) {
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    int u_index = parameters.field_indexes.u;
    int v_index = parameters.field_indexes.v;
    int w_index = parameters.field_indexes.w;
    int T_index = parameters.field_indexes.T;
    int im1, ip1, jm1, jp1;
    double dx = parameters.dx;
    double dy = parameters.dy;
    double dz = parameters.dz;
    double dt = parameters.dt;
    double T_inf = parameters.T_inf;
    double rho_inf = parameters.rho_inf;
    double u_ijk, u_ip1jk, u_im1jk, u_iphjk, u_imhjk;
    double v_ijk, v_ijp1k, v_ijm1k, v_ijphk, v_ijmhk;
    double w_ijk, w_ijkp1, w_ijkm1, w_ijkp2, w_ijkm2, w_ijkph, w_ijkmh;
    double T_ijk, T_im1jk, T_ip1jk, T_ijm1k, T_ijp1k, T_ijkp1, T_ijkp2, T_ijkm1, T_ijkm2;
    double p_ijk, p_im1jk, p_ip1jk, p_ijm1k, p_ijp1k, p_ijkp1, p_ijkp2, p_ijkm1, p_ijkm2;
    double rho_ijk, rho_im1jk, rho_ip1jk, rho_ijm1k, rho_ijp1k, rho_ijkp1, rho_ijkp2, rho_ijkm1, rho_ijkm2;
    double ux, vy, wz, f;
    double rho, rhox, rhoy, rhoz, px, py, pz;
    // Loop over nodes to compute f
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    int size = Nx * Ny * Nz;
    for (int ijk = idx; ijk < size; ijk += stride) {
        int i = ijk / (Ny * Nz);
        int j = (ijk % (Ny * Nz)) / Nz;
        int k = ijk % Nz;               
        // Indexes for periodic boundary conditions
        im1 = (i - 1 + Nx - 1) % (Nx - 1);
        jm1 = (j - 1 + Ny - 1) % (Ny - 1);
        ip1 = (i + 1) % (Nx - 1);
        jp1 = (j + 1) % (Ny - 1);                
        // Local nodes
        u_ijk = U[u_index + IDX(i, j, k, Nx, Ny, Nz)];
        v_ijk = U[v_index + IDX(i, j, k, Nx, Ny, Nz)];
        w_ijk = U[w_index + IDX(i, j, k, Nx, Ny, Nz)];
        T_ijk = U[T_index + IDX(i, j, k, Nx, Ny, Nz)];
        p_ijk = p[IDX(i, j, k, Nx, Ny, Nz)];
        rho_ijk = T_inf * rho_inf / T_ijk;
        // Periodic boundary conditions on xy
        u_im1jk = U[u_index + IDX(im1, j, k, Nx, Ny, Nz)];
        u_ip1jk = U[u_index + IDX(ip1, j, k, Nx, Ny, Nz)];
        v_ijm1k = U[v_index + IDX(i, jm1, k, Nx, Ny, Nz)];
        v_ijp1k = U[v_index + IDX(i, jp1, k, Nx, Ny, Nz)];
        T_im1jk = U[T_index + IDX(im1, j, k, Nx, Ny, Nz)];
        T_ip1jk = U[T_index + IDX(ip1, j, k, Nx, Ny, Nz)];
        T_ijm1k = U[T_index + IDX(i, jm1, k, Nx, Ny, Nz)];
        T_ijp1k = U[T_index + IDX(i, jp1, k, Nx, Ny, Nz)];
        p_im1jk = p[IDX(im1, j, k, Nx, Ny, Nz)];
        p_ip1jk = p[IDX(ip1, j, k, Nx, Ny, Nz)];
        p_ijm1k = p[IDX(i, jm1, k, Nx, Ny, Nz)];
        p_ijp1k = p[IDX(i, jp1, k, Nx, Ny, Nz)];
        rho_im1jk = T_inf * rho_inf / T_im1jk;
        rho_ip1jk = T_inf * rho_inf / T_ip1jk;
        rho_ijm1k = T_inf * rho_inf / T_ijm1k;
        rho_ijp1k = T_inf * rho_inf / T_ijp1k;
        // dw/dz 
        if (k == 0) { // Bottom boundary                    
            w_ijkp1 = U[w_index + IDX(i, j, k + 1, Nx, Ny, Nz)];
            w_ijkp2 = U[w_index + IDX(i, j, k + 2, Nx, Ny, Nz)];
            T_ijkp1 = U[T_index + IDX(i, j, k + 1, Nx, Ny, Nz)];
            T_ijkp2 = U[T_index + IDX(i, j, k + 2, Nx, Ny, Nz)];
            p_ijkp1 = p[IDX(i, j, k + 1, Nx, Ny, Nz)];
            p_ijkp2 = p[IDX(i, j, k + 2, Nx, Ny, Nz)];
            rho_ijkp1 = T_inf * rho_inf / T_ijkp1;
            rho_ijkp2 = T_inf * rho_inf / T_ijkp2;
            wz = (-3 * w_ijk + 4 * w_ijkp1 - w_ijkp2) / (2 * dz); // dw/dz at z = z_min
            rhoz = (-3 * rho_ijk + 4 * rho_ijkp1 - rho_ijkp2) / (2 * dz);
            pz = (-3 * p_ijk + 4 * p_ijkp1 - p_ijkp2) / (2 * dz);
        } else if (k == Nz - 1) { // Top boundary
            w_ijkm1 = U[w_index + IDX(i, j, k - 1, Nx, Ny, Nz)];
            w_ijkm2 = U[w_index + IDX(i, j, k - 2, Nx, Ny, Nz)];
            T_ijkm1 = U[T_index + IDX(i, j, k - 1, Nx, Ny, Nz)];
            T_ijkm2 = U[T_index + IDX(i, j, k - 2, Nx, Ny, Nz)];
            p_ijkm1 = p[IDX(i, j, k - 1, Nx, Ny, Nz)];
            p_ijkm2 = p[IDX(i, j, k - 2, Nx, Ny, Nz)];
            rho_ijkm1 = T_inf * rho_inf / T_ijkm1;
            rho_ijkm2 = T_inf * rho_inf / T_ijkm2;
            wz = (3 * w_ijk - 4 * w_ijkm1 + w_ijkm2) / (2 * dz); // dw/dz at z = z_max
            rhoz = (3 * rho_ijk - 4 * rho_ijkm1 + rho_ijkm2) / (2 * dz);
            pz = (3 * p_ijk - 4 * p_ijkm1 + p_ijkm2) / (2 * dz);
        } else { // Interior
            w_ijkp1 = U[w_index + IDX(i, j, k + 1, Nx, Ny, Nz)];
            w_ijkm1 = U[w_index + IDX(i, j, k - 1, Nx, Ny, Nz)];
            T_ijkp1 = U[T_index + IDX(i, j, k + 1, Nx, Ny, Nz)];
            T_ijkm1 = U[T_index + IDX(i, j, k - 1, Nx, Ny, Nz)];
            rho_ijkp1 = T_inf * rho_inf / T_ijkp1;
            rho_ijkm1 = T_inf * rho_inf / T_ijkm1;
            p_ijkp1 = p[IDX(i, j, k + 1, Nx, Ny, Nz)];
            p_ijkm1 = p[IDX(i, j, k - 1, Nx, Ny, Nz)];
            w_ijkph = 0.5 * (w_ijk + w_ijkp1);
            w_ijkmh = 0.5 * (w_ijk + w_ijkm1);
            wz = (w_ijkph - w_ijkmh) / dz; // dw/dz at z = z_k
            rhoz = (rho_ijkp1 - rho_ijkm1) / (2 * dz); // drho/dz at z = z_k
            pz = (p_ijkp1 - p_ijkm1) / (2 * dz); // dp/dz at z = z_k
        }
        // Half derivatives
        u_iphjk = 0.5 * (u_ip1jk + u_ijk);
        u_imhjk = 0.5 * (u_ijk + u_im1jk);
        v_ijphk = 0.5 * (v_ijp1k + v_ijk);
        v_ijmhk = 0.5 * (v_ijk + v_ijm1k);
        ux = (u_iphjk - u_imhjk) / dx; // du/dx
        vy = (v_ijphk - v_ijmhk) / dy; // dv/dy
        // Density and pressure gradients using central differences
        rhox = (rho_ip1jk - rho_im1jk) / (2 * dx);
        rhoy = (rho_ijp1k - rho_ijm1k) / (2 * dy);
        px = (p_ip1jk - p_im1jk) / (2 * dx);
        py = (p_ijp1k - p_ijm1k) / (2 * dy);
        if (i < Nx - 1 && j < Ny - 1 && k < Nz - 1) {
            // Compute rho / dt * div(U) and store it for many DFT (contiguous z slices)
            rho = T_inf * rho_inf / T_ijk;
            f = rho * (ux + vy + wz) / dt + (rhox * px + rhoy * py + rhoz * pz) / rho;
            f_in[FFTWIDX(i, j, k, Nx - 1, Ny - 1, Nz - 1)] = make_hipDoubleComplex(f, 0.0);
        }
        // Fill p_top
        if (k == Nz - 1 && j < Ny - 1 && i < Nx - 1) {
            p_top_in[FFTWIDX(i, j, 0, Nx - 1, Ny - 1, Nz - 1)] = make_hipDoubleComplex(p[IDX(i, j, k, Nx, Ny, Nz)], 0.0);
        }
    }
}

__global__ 
void update_coefficients(double *gamma, double *b,  hipfftDoubleComplex *d, hipfftDoubleComplex *f_out, hipfftDoubleComplex *p_top_out, Parameters parameters) {
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    double dz = parameters.dz;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    int size = (Nx - 1) * (Ny - 1) * (Nz - 1);
    for (int rsk = idx; rsk < size; rsk += stride) {
        int r = rsk / ((Ny - 1) * (Nz - 1));
        int s = (rsk % ((Ny - 1) * (Nz - 1))) / (Nz - 1);
        int k = rsk % (Nz - 1);
        if (k == 0) {
            // First equation k=0
            d[IDX(r, s, k, Nx - 1, Ny - 1, Nz - 1)] = hipCmul(make_hipDoubleComplex(0.5 * dz, 0.0), f_out[FFTWIDX(r, s, 1, Nx - 1, Ny - 1, Nz - 1)]);
            // Last equation k=Nz-2
            d[IDX(r, s, Nz - 2, Nx - 1, Ny - 1, Nz - 1)] = hipCsub(f_out[FFTWIDX(r, s, Nz - 2, Nx - 1, Ny - 1, Nz - 1)], hipCdiv(p_top_out[FFTWIDX(r, s, 0, Nx - 1, Ny - 1, Nz - 1)], make_hipDoubleComplex(dz * dz, 0.0)));
        } else {
            if (k < Nz - 1) {
                // b[IDX(r, s, k, Nx - 1, Ny - 1, Nz - 1)] = gamma[FFTWIDX(r, s, 0, Nx - 1, Ny - 1, 0)] / (dz * dz);
                b[IDX(r, s, k, Nx - 1, Ny - 1, Nz - 1)] = gamma[IDX(r, s, 0, Nx - 1, Ny - 1, 1)] / (dz * dz);
                if (k < Nz - 2)
                    d[IDX(r, s, k, Nx - 1, Ny - 1, Nz - 1)] = f_out[FFTWIDX(r, s, k, Nx - 1, Ny - 1, Nz - 1)];
            }
        }
    }
}

__global__
void post_fft(double *p, hipfftDoubleComplex *p_out, Parameters parameters) {
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    // Loop over nodes to compute f
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    int size = Nx * Ny * Nz;
    for (int ijk = idx; ijk < size; ijk += stride) {
        int i = ijk / (Ny * Nz);
        int j = (ijk % (Ny * Nz)) / Nz;
        int k = ijk % Nz;  
        if (i < Nx - 1 && j < Ny - 1 && k < Nz - 1) {
            p[IDX(i, j, k, Nx, Ny, Nz)] = hipCreal(hipCdiv(p_out[FFTWIDX(i, j, k, Nx - 1, Ny - 1, Nz - 1)], make_hipDoubleComplex((Nx - 1) * (Ny - 1), 0.0)));
        }
        // Periodic boundary conditions on xy
        if (i == Nx - 1) { // Right boundary
            p[IDX(i, j, k, Nx, Ny, Nz)] = p[IDX(0, j, k, Nx, Ny, Nz)];
        }
        if (j == Ny - 1) { // Front boundary
            p[IDX(i, j, k, Nx, Ny, Nz)] = p[IDX(i, 0, k, Nx, Ny, Nz)];
        }
    }
}

void solve_pressure(double *U, double *p, double *gamma, double *a, double *b, double *c, hipfftDoubleComplex *d, hipfftDoubleComplex *l, hipfftDoubleComplex *u, hipfftDoubleComplex *y, hipfftDoubleComplex *data_in, hipfftDoubleComplex *data_out, hipfftDoubleComplex *p_top_in, hipfftDoubleComplex *p_top_out, Parameters parameters) {
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    int n[] = {Nx - 1, Ny - 1};
    int idist = (Nx - 1) * (Ny - 1);
    int odist = (Nx - 1) * (Ny - 1);
    int istride = 1;
    int ostride = 1;
    int howmany = Nz - 1;
    int *inembed = NULL;
    int *onembed = NULL;
    hipfftHandle p_plan = 0, f_plan = 0, p_top_plan = 0;

    // Compute f = rho / dt * div(U)
    if (parameters.variable_density == 0)
        compute_f<<<BLOCKS, THREADS>>>(U, data_in, p_top_in, p, parameters);
    else
        compute_f_density<<<BLOCKS, THREADS>>>(U, data_in, p_top_in, p, parameters);
    checkCuda(hipGetLastError());
    
    // Plans for FFT2D
    CHECK_CUFFT(hipfftPlanMany(&p_plan, 2, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, howmany)); // FFT2(f_k) for each z slice
    CHECK_CUFFT(hipfftPlan2d(&p_top_plan, Nx - 1, Ny - 1, HIPFFT_Z2Z)); // FFT2(p_top)

    // Compute FFT2D
    CHECK_CUFFT(hipfftExecZ2Z(p_top_plan, p_top_in, p_top_out, HIPFFT_FORWARD)); // FFT2(p_top)
    CHECK_CUFFT(hipfftExecZ2Z(p_plan, data_in, data_out, HIPFFT_FORWARD)); // FFT2D(f_k) for each z slice
    CHECK(hipDeviceSynchronize());

    // Update coefficients, including f in pseudo-Fourier space
    update_coefficients<<<BLOCKS, THREADS>>>(gamma, b, d, data_out, p_top_out, parameters);
    checkCuda(hipGetLastError());

    // Compute r,s systems of equations using thomas algorithm
    thomas_algorithm<<<BLOCKS, THREADS>>>(a, b, c, d, data_in, l, u, y, parameters);
    checkCuda(hipGetLastError());

    // Compute IFFT2D
    CHECK_CUFFT(hipfftExecZ2Z(p_plan, data_in, data_out, HIPFFT_BACKWARD));
    CHECK(hipDeviceSynchronize());

    // Post FFT
    post_fft<<<BLOCKS, THREADS>>>(p, data_out, parameters);
    checkCuda(hipGetLastError());

    // Destroy plans
    hipfftDestroy(p_top_plan);
    hipfftDestroy(p_plan);
    hipfftDestroy(f_plan);
}

void solve_pressure_iterative(double *U, double *p, double *gamma, double *a, double *b, double *c, hipfftDoubleComplex *d, hipfftDoubleComplex *l, hipfftDoubleComplex *u, hipfftDoubleComplex *y, hipfftDoubleComplex *data_in, hipfftDoubleComplex *data_out, hipfftDoubleComplex *p_top_in, hipfftDoubleComplex *p_top_out, Parameters parameters, double *error, int *max_iter) {
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    int size = Nx * Ny * Nz;
    double h_tol, *d_tol; 
    char solver_log_message[128];
    // Create a temporary array on GPU to store the pressure field
    double *p_tmp;
    int m;
    checkCuda(hipMalloc((void **)&p_tmp, size * sizeof(double)));
    // Allocate memory for the tolerance
    checkCuda(hipMalloc((void **)&d_tol, sizeof(double)));
    for (m = 0; m < parameters.pressure_solver_iter; m++) {
        // Copy the initial pressure field to the temporary array
        checkCuda(hipMemcpy(p_tmp, p, size * sizeof(double), hipMemcpyDeviceToDevice));
        solve_pressure(U, p, gamma, a, b, c, d, l, u, y, data_in, data_out, p_top_in, p_top_out, parameters);
        norm<<<BLOCKS, THREADS>>>(p, p_tmp, d_tol, INFINITY, size);
        checkCuda(hipGetLastError());
        checkCuda(hipMemcpy(&h_tol, d_tol, sizeof(double), hipMemcpyDeviceToHost));
        if (h_tol <= parameters.pressure_solver_tol) {
            break;
        }        
    }
    if (parameters.pressure_solver_log == 1) {
        sprintf(solver_log_message, "Pressure solver: Error = %e, iterations = %d", h_tol, m);
        log_message(parameters, solver_log_message);
    }
    *max_iter = m;
    *error = h_tol;
    // Free memory
    checkCuda(hipFree(p_tmp));
    checkCuda(hipFree(d_tol));
}