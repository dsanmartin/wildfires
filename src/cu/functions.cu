#include "hip/hip_runtime.h"
/**
 * @file functions.c
 * @brief Implementation of various functions used in the wildfire simulation.
 * @author Daniel San Martin (dsanmartinreyes@gmail.com)
 */

#include "../../include/cu/functions.cuh"

double power_law(double z, double u_r, double z_r, double alpha_u) {
    return u_r * pow(z / z_r, alpha_u);
}

double gaussian(double x, double y, double z, double x_0, double y_0, double z_0, double sx, double sy, double sz) {
    return exp(-pow((x - x_0) / sx, 2.0) - pow((y - y_0) / sy, 2.0) - pow((z - z_0) / sz, 2.0));
}

double K(double T, double A, double T_a) {
    return A * exp(-T_a / T);
}

double H(double x, double T_pc) {
    if (x > T_pc) {
        return 1.0;
    } else {
        return 0.0;
    }
}

double f_damping(double z, double u_tau, double nu) {
    return 1 - exp(-z * u_tau / 25 / nu);
}

double source(double T, double Y, double H_R, double A, double T_a, double h, double a_v, double T_inf, double c_p, double rho, double T_pc) {
    return H_R * Y * K(T, A, T_a) * H(T, T_pc) / c_p - h * a_v * (T - T_inf) / (c_p * rho);
}

void timestep_reports(double *y_n, double *CFL, double *Y_min, double *Y_max, double *T_min, double *T_max, Parameters parameters) {
    // printf("Timestep reports\n");
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    int Nz_Y_max = parameters.Nz_Y_max;
    int u_index = parameters.field_indexes.u;
    int v_index = parameters.field_indexes.v;
    int w_index = parameters.field_indexes.w;
    int T_index = parameters.field_indexes.T;
    int Y_index = parameters.field_indexes.Y;
    // int size = Nx * Ny * Nz;
    double dx = parameters.dx;
    double dy = parameters.dy;
    double dz = parameters.dz;
    double dt = parameters.dt;
    double max_u = 0.0;
    double max_v = 0.0;
    double max_w = 0.0;
    double abs_u, abs_v, abs_w;
    // double CFL_tmp = 0.0;
    double Y_min_tmp = 0.0, Y_max_tmp = -1e9;
    double T_min_tmp = 1e9, T_max_tmp = -1e9;
    // int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // int stride = gridDim.x * blockDim.x;
    // for (int ijk = idx; ijk < size; ijk += stride) {
        // int i = ijk / (Ny * Nz);
        // int j = (ijk % (Ny * Nz)) / Nz;
        // int k = ijk % Nz;
    for (int i = 0; i < Nx; i++) {
        for (int j = 0; j < Ny; j++) {
            for (int k = 0; k < Nz; k++) {
                abs_u = fabs(y_n[u_index + IDX(i, j, k, Nx, Ny, Nz)]);
                abs_v = fabs(y_n[v_index + IDX(i, j, k, Nx, Ny, Nz)]);
                abs_w = fabs(y_n[w_index + IDX(i, j, k, Nx, Ny, Nz)]);
                max_u = MAX(max_u, abs_u);
                max_v = MAX(max_v, abs_v);
                max_w = MAX(max_w, abs_w);
                if (k < Nz_Y_max) {
                    Y_min_tmp = MIN(Y_min_tmp, y_n[Y_index + IDX(i, j, k, Nx, Ny, Nz_Y_max)]);
                    Y_max_tmp = MAX(Y_max_tmp, y_n[Y_index + IDX(i, j, k, Nx, Ny, Nz_Y_max)]);
                }
                T_min_tmp = MIN(T_min_tmp, y_n[T_index + IDX(i, j, k, Nx, Ny, Nz)]);
                T_max_tmp = MAX(T_max_tmp, y_n[T_index + IDX(i, j, k, Nx, Ny, Nz)]);
            }
        }
    }
    *CFL = dt * (max_u / dx + max_v / dy + max_w / dz);
    *Y_min = Y_min_tmp;
    *Y_max = Y_max_tmp;
    *T_min = T_min_tmp;
    *T_max = T_max_tmp;
}

void initial_conditions(double *u, double *v, double *w, double *T, double *Y, double *p, Parameters parameters) {
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    int Nz_Y_max = parameters.Nz_Y_max;
    int *Nz_Y = parameters.Nz_Y;
    /* Velocity parameters */
    double u_r = parameters.u_r;
    double z_r = parameters.z_r;
    double alpha_u = parameters.alpha_u;
    /* Temperature parameters */
    double x_0 = parameters.T0_x_center;
    double y_0 = parameters.T0_y_center;
    double z_0 = parameters.T0_z_center;
    double sx = parameters.T0_length;
    double sy = parameters.T0_width;
    double sz = parameters.T0_height;
    double T_hot = parameters.T_hot;
    double T_inf = parameters.T_inf;
    /* Spatial domain */
    double *x = parameters.x;
    double *y = parameters.y;
    double *z = parameters.z;
    /* Pressure paramenters */
    double p_top = parameters.p_top;
    /* Fill arrays */
    for (int i = 0; i < Nx; i++) {
        for (int j = 0; j < Ny; j++) {
            for (int k = 0; k < Nz; k++) {
                u[IDX(i, j, k, Nx, Ny, Nz)] = power_law(z[k], u_r, z_r, alpha_u);
                v[IDX(i, j, k, Nx, Ny, Nz)] = 0.0;
                w[IDX(i, j, k, Nx, Ny, Nz)] = 0.0;
                T[IDX(i, j, k, Nx, Ny, Nz)] = T_inf +  (T_hot - T_inf) * gaussian(x[i], y[j], z[k], x_0, y_0, z_0, sx, sy, sz);
                if (k == Nz - 1) {
                    p[IDX(i, j, k, Nx, Ny, Nz)] = p_top;
                } else {
                    p[IDX(i, j, k, Nx, Ny, Nz)] = 0.0;
                }
                if (k < Nz_Y[IDX(i, j, 0, Nx, Ny, 1)]) {
                    Y[IDX(i, j, k, Nx, Ny, Nz_Y_max)] = 1.0;
                }
            }
        }
    }
}