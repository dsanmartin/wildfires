#include "hip/hip_runtime.h"
/**
 * @file solver.cu
 * @author Daniel San Martin (dsanmartinreyes@gmail.com)
 * @brief Functions for solving the partial differential equations of the wildfire simulation.
 * @version 0.1
 * @date 2024-07-21
 * 
 * @copyright Copyright (c) 2024
 * 
 */

#include "../../include/cu/solver.cuh"

__global__ 
void euler_step(double dt, double *y_n, double *y_np1, double *F, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;    
    for (int i = idx; i < size; i += stride) {
        y_np1[i] = y_n[i] + dt * F[i];
    }
}

__global__
void RK2_step(double dt, double *y_n, double *y_np1, double *k1, double *k2, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = idx; i < size; i += stride) {
        y_np1[i] = y_n[i] + 0.5 * dt * (k1[i] + k2[i]);
    }
}

__global__
void RK4_step(double dt, double *y_n, double *y_np1, double *k1, double *k2, double *k3, double *k4, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = idx; i < size; i += stride) {
        y_np1[i] = y_n[i] +  (dt / 6) * (k1[i] + 2 * k2[i] + 2 * k3[i] + k4[i]);
    }
}

void create_y_0(double *u, double *v, double *w, double *T, double *Y, double *y_0, Parameters parameters) {
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    int Nz_Y_max = parameters.Nz_Y_max;
    int size = Nx * Ny * Nz;
    int size_Y = Nx * Ny * Nz_Y_max;
    for (int i = 0; i < size; i++) {
        y_0[parameters.field_indexes.u + i] = u[i];
        y_0[parameters.field_indexes.v + i] = v[i];
        y_0[parameters.field_indexes.w + i] = w[i];
        y_0[parameters.field_indexes.T + i] = T[i];
        if (i < size_Y) {
            y_0[parameters.field_indexes.Y + i] = Y[i];
        }
    }
}

void euler(double t_n, double *y_n, double *y_np1, double *F, double *U_turbulence, double *z, double *z_ibm, int *Nz_Y, int *cut_nodes, double dt, int size, Parameters parameters) {   
    Phi(t_n, y_n, F, U_turbulence, z, z_ibm, Nz_Y, cut_nodes, parameters);
    euler_step<<<BLOCKS, THREADS>>>(dt, y_n, y_np1, F, size);
    hipDeviceSynchronize();
}

void RK2(double t_n, double *y_n, double *y_np1, double *k, double *F, double *U_turbulence, double *z, double *z_ibm, int *Nz_Y, int *cut_nodes, double dt, int size, Parameters parameters) {
    int k1_index = 0;
    int k2_index = size;
    Phi(t_n, y_n, k + k1_index, U_turbulence, z, z_ibm, Nz_Y, cut_nodes, parameters);
    caxpy<<<BLOCKS, THREADS>>>(F, k + k1_index, y_n, dt, size);
    hipDeviceSynchronize();
    Phi(t_n + dt, F, k + k2_index, U_turbulence, z, z_ibm, Nz_Y, cut_nodes, parameters);
    RK2_step<<<BLOCKS, THREADS>>>(dt, y_n, y_np1, k + k1_index, k + k2_index, size);
    hipDeviceSynchronize();
}

void RK4(double t_n, double *y_n, double *y_np1, double *k, double *F, double *U_turbulence, double *z, double *z_ibm, int *Nz_Y, int *cut_nodes, double dt, int size, Parameters parameters) {
    int k1_index = 0;
    int k2_index = size;
    int k3_index = 2 * size;
    int k4_index = 3 * size;
    Phi(t_n, y_n, k + k1_index, U_turbulence, z, z_ibm, Nz_Y, cut_nodes, parameters);
    caxpy<<<BLOCKS, THREADS>>>(F, k + k1_index, y_n, dt * 0.5, size);
    hipDeviceSynchronize();
    Phi(t_n + 0.5 * dt, F, k + k2_index, U_turbulence, z, z_ibm, Nz_Y, cut_nodes, parameters);
    caxpy<<<BLOCKS, THREADS>>>(F, k + k2_index, y_n, dt * 0.5, size);
    hipDeviceSynchronize();
    Phi(t_n + 0.5 * dt, F, k + k3_index, U_turbulence, z, z_ibm, Nz_Y, cut_nodes, parameters);
    caxpy<<<BLOCKS, THREADS>>>(F, k + k3_index, y_n, dt, size);
    hipDeviceSynchronize();
    Phi(t_n + dt, F, k + k4_index, U_turbulence, z, z_ibm, Nz_Y, cut_nodes, parameters);
    RK4_step<<<BLOCKS, THREADS>>>(dt, y_n, y_np1, k + k1_index, k + k2_index, k + k3_index, k + k4_index, size);
    hipDeviceSynchronize();
}

void solve_PDE(double *y_n, double *p, Parameters parameters) {
    int Nx = parameters.Nx;
    int Ny = parameters.Ny;
    int Nz = parameters.Nz;
    int Nt = parameters.Nt;
    int NT = parameters.NT;
    int Nz_Y_max = parameters.Nz_Y_max;
    int size = 4 * Nx * Ny * Nz + Nx * Ny * Nz_Y_max;
    int n_save;
    int k_size = (strncmp(parameters.method, "RK4", 3) == 0) ? 4 : 2;
    double step_time, solver_time, cum_step_time = 0.0;
    double CFL = 0.0, T_min = 1e9, T_max = -1e9, Y_min = 1e9, Y_max = -1e9;
    double error;
    int max_iter;
    double *t = parameters.t;
    double dt = parameters.dt;
    // Host data
    double *y_np1_host, *p_host;
    // Device data
    double *d_x, *d_y, *d_z, *y_np1, *F, *k, *R_turbulence, *z_ibm, *kx, *ky, *gamma;
    int *Nz_Y, *cut_nodes;
    // Arrays for pressure Poisson Problema
    double *a, *b, *c;
    hipfftDoubleComplex *d, *l, *u, *y;
    hipfftDoubleComplex *data_in, *data_out, *p_top_in, *p_top_out;
    // clock_t start, end, step_start, step_end; // Timers
    struct timeval start_solver, end_solver, start_ts, end_ts; // Timers
    // Messages
    char solver_time_message[128];
    char pressure_log_message[128];
    char formatted_time[64];
    // Host memory allocation
    y_np1_host = (double *) malloc(size * sizeof(double));
    p_host = (double *) malloc(Nx * Ny * Nz * sizeof(double));
    // Memory allocation for device data
    CHECK(hipMalloc((void **)&d_x, Nx * sizeof(double)));
    CHECK(hipMalloc((void **)&d_y, Ny * sizeof(double)));
    CHECK(hipMalloc((void **)&d_z, Nz * sizeof(double)));
    CHECK(hipMalloc((void **)&y_np1, size * sizeof(double)));
    CHECK(hipMalloc((void **)&F, size * sizeof(double)));
    CHECK(hipMalloc((void **)&k, k_size * size * sizeof(double)));
    CHECK(hipMalloc((void **)&R_turbulence, 25 * Nx * Ny * Nz * sizeof(double)));
    CHECK(hipMalloc((void **)&z_ibm, Nx * Ny * Nz * sizeof(double)));
    CHECK(hipMalloc((void **)&Nz_Y, Nx * Ny * sizeof(int)));
    CHECK(hipMalloc((void **)&cut_nodes, Nx * Ny * sizeof(int)));
    CHECK(hipMalloc((void **)&kx, (Nx - 1) * sizeof(double)));
    CHECK(hipMalloc((void **)&ky, (Ny - 1) * sizeof(double)));
    CHECK(hipMalloc((void **)&gamma, (Nx - 1) * (Ny - 1) * (Nz - 1) * sizeof(double)));
    // Allocate memory for Poisson problem
    CHECK(hipMalloc((void **)&a, (Nx - 1) * (Ny - 1) * (Nz - 2) * sizeof(double)));
    CHECK(hipMalloc((void **)&b, (Nx - 1) * (Ny - 1) * (Nz - 1) * sizeof(double)));
    CHECK(hipMalloc((void **)&c, (Nx - 1) * (Ny - 1) * (Nz - 2) * sizeof(double)));
    CHECK(hipMalloc((void **)&d, (Nx - 1) * (Ny - 1) * (Nz - 1) * sizeof(hipfftDoubleComplex)));
    CHECK(hipMalloc((void **)&l, (Nx - 1) * (Ny - 1) * (Nz - 2) * sizeof(hipfftDoubleComplex)));
    CHECK(hipMalloc((void **)&u, (Nx - 1) * (Ny - 1) * (Nz - 1) * sizeof(hipfftDoubleComplex)));
    CHECK(hipMalloc((void **)&y, (Nx - 1) * (Ny - 1) * (Nz - 1) * sizeof(hipfftDoubleComplex)));
    CHECK(hipMalloc((void **)&p_top_in, (Nx - 1) * (Ny - 1) * sizeof(hipfftDoubleComplex)));
    CHECK(hipMalloc((void **)&p_top_out, (Nx - 1) * (Ny - 1) * sizeof(hipfftDoubleComplex)));
    CHECK(hipMalloc((void **)&data_in, (Nx - 1) * (Ny - 1) * (Nz - 1) * sizeof(hipfftDoubleComplex)));
    CHECK(hipMalloc((void **)&data_out, (Nx - 1) * (Ny - 1) * (Nz - 1) * sizeof(hipfftDoubleComplex)));
    // Copy host data to device
    CHECK(hipMemcpy(d_x, parameters.x, Nx * sizeof(double), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, parameters.y, Ny * sizeof(double), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_z, parameters.z, Nz * sizeof(double), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(z_ibm, parameters.z_ibm, Nx * Ny * Nz * sizeof(double), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(kx, parameters.kx, (Nx - 1) * sizeof(double), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(ky, parameters.ky, (Ny - 1) * sizeof(double), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(Nz_Y, parameters.Nz_Y, Nx * Ny * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(cut_nodes, parameters.cut_nodes, Nx * Ny * sizeof(int), hipMemcpyHostToDevice));
    // Fill gamma and coefficients
    gammas_and_coefficients<<<BLOCKS, THREADS>>>(kx, ky, gamma, a, b, c, d_z, parameters);
    checkCuda(hipGetLastError());
    CHECK(hipDeviceSynchronize());
    // Solver time
    // start = clock();
    gettimeofday(&start_solver, NULL);
    // Time integration
    for (int n = 1; n <= Nt; n++) { 
        // step_start = clock(); // Start step timer
        gettimeofday(&start_ts, NULL);
        // Compute U^*, T^{n+1}, Y^{n+1}
        // Check time integration method
        if (strncmp(parameters.method, "Euler", 5) == 0) {
            euler(t[n], y_n, y_np1, F, R_turbulence, d_z, z_ibm, Nz_Y, cut_nodes, dt, size, parameters);
        } else if (strncmp(parameters.method, "RK2", 3) == 0) {
            RK2(t[n], y_n, y_np1, k, F, R_turbulence, d_z, z_ibm, Nz_Y, cut_nodes, dt, size, parameters);
        } else if (strncmp(parameters.method, "RK4", 3) == 0) {
            RK4(t[n], y_n, y_np1, k, F, R_turbulence, d_z, z_ibm, Nz_Y, cut_nodes, dt, size, parameters);
        } else {
            log_message(parameters, "Time integration method not found.");
            exit(1);
        }  
        // Solve Poisson problem for pressure 
        if (parameters.variable_density == 0) { // Constant density, direct solver
            solve_pressure(y_np1, p, d_z, gamma, a, b, c, d, l, u, y, data_in, data_out, p_top_in, p_top_out, parameters);            
        } else { // Variable density, iterative solver
            solve_pressure_iterative(y_np1, p, d_z, gamma, a, b, c, d, l, u, y, data_in, data_out, p_top_in, p_top_out, parameters, &error, &max_iter);
        }
        checkCuda(hipGetLastError());
        // Chorin's projection method        
        velocity_correction<<<BLOCKS, THREADS>>>(y_np1, p, d_z, 1, parameters);
        checkCuda(hipGetLastError());
        // Boundary conditions
        boundary_conditions<<<BLOCKS, THREADS>>>(y_np1, d_z, Nz_Y, cut_nodes, parameters);
        checkCuda(hipGetLastError());
        // Bounds
        bounds<<<BLOCKS, THREADS>>>(y_np1, parameters);
        checkCuda(hipGetLastError());
        // Add source when t_n <= t_source
        if (t[n] <= parameters.t_source) {
            temperature_source<<<BLOCKS, THREADS>>>(d_x, d_y, d_z, y_np1, parameters);
            checkCuda(hipGetLastError());
        }
        // End step timer
        // step_end = clock(); 
        gettimeofday(&end_ts, NULL);
        // Compute step time
        // step_time = (double) (step_end - step_start) / CLOCKS_PER_SEC;
        step_time = ((end_ts.tv_sec  - start_ts.tv_sec) * 1000000u + end_ts.tv_usec - start_ts.tv_usec) / 1.e6;        
        // Show time step and average time each 100 steps
        cum_step_time += step_time;
        if (n % 100 == 0) {
            printf("Time step: %d, Average time: %lf s\n", n, cum_step_time / n);
        }
        // Save data each NT steps and at the last step
        if (n % NT == 0 || n == Nt) {  
            n_save = n / NT;
            // Copy y_np1 and p to host
            hipMemcpy(y_np1_host, y_np1, size * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(p_host, p, Nx * Ny * Nz * sizeof(double), hipMemcpyDeviceToHost);
            timestep_reports(y_np1_host, &CFL, &Y_min, &Y_max, &T_min, &T_max, parameters);
            log_timestep(parameters, n, t[n], step_time, CFL, T_min, T_max, Y_min, Y_max);
            if (parameters.variable_density == 1) {
                sprintf(pressure_log_message, "Pressure solver: Error = %e, iterations = %d", error, max_iter);
                log_message(parameters, pressure_log_message);
            }            
            save_data(y_np1_host, p_host, n_save, parameters);
        }
        // Update y_n
        copy<<<BLOCKS, THREADS>>>(y_n, y_np1, size);
        checkCuda(hipGetLastError());
    }
    // end = clock();
    gettimeofday(&end_solver, NULL);
    // printf("Solver time: %lf s\n", (double) (end - start) / CLOCKS_PER_SEC);
    solver_time = ((end_solver.tv_sec  - start_solver.tv_sec) * 1000000u + end_solver.tv_usec - start_solver.tv_usec) / 1.e6;
    format_seconds(solver_time, formatted_time);
    // Create the solver time using format hh:mm:ss s
    sprintf(solver_time_message, "\nSolver time: %s", formatted_time);
    log_message(parameters, solver_time_message);
    // Free memory
    // Host memory
    free(y_np1_host);
    free(p_host);
    // Device memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    hipFree(y_np1);
    hipFree(F);
    hipFree(R_turbulence);
    hipFree(k);
    // Free memory for Poisson problem
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(d);
    hipFree(l);
    hipFree(u);
    hipFree(y);
    hipFree(p_top_in);
    hipFree(p_top_out);
    hipFree(data_in);
    hipFree(data_out);
}
